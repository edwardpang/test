#include <stdio.h>

int main (void)
{
	hipDeviceProp_t	prop;

	int count;

	hipGetDeviceCount (&count);
	for (int i=0; i<count; i++)
	{
		hipGetDeviceProperties (&prop, i);
		printf ("General Information for device %d\n", i);
		printf ("Name :\t\t\t%s\n", prop.name);
		printf ("CUDA Ver :\t\t%d.%d\n", prop.major, prop.minor);
		printf ("Clock rate :\t\t%d\n", prop.clockRate);

		printf ("Max Threads Per Block :\t%d\n", prop.maxThreadsPerBlock);
		printf ("Max Threads Dim :\t[%d %d %d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf ("Max Grid Size :\t\t[%d %d %d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	}

	return 0;
}

